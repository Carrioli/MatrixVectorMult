#include "hip/hip_runtime.h"
// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

__global__ void MatrixMulCUDA(double* C, double* A, double* B) {
    int index = threadIdx.x;

    __shared__ double shared_b[1024];
    shared_b[index] = B[index];
    __syncthreads();
    for (int i = 0; i < 1024; i++)
        C[index] += (A[1024 * index + i] * shared_b[i]);
    __syncthreads();
}

void ConstantInit(double* data, int size) {
    for (int i = 0; i < size; ++i) {
        data[i] = i;
    }
}

/**
 * Run a simple test of matrix multiplication using CUDA
 */
int MatrixMultiply(const dim3& dimsA, const dim3& dimsB) {
    // Allocate host memory for matrices A and B
    unsigned int size_A = dimsA.x * dimsA.y;
    unsigned int mem_size_A = sizeof(double) * size_A;
    double* h_A;
    checkCudaErrors(hipHostMalloc(&h_A, mem_size_A));
    unsigned int size_B = dimsB.x * dimsB.y;
    unsigned int mem_size_B = sizeof(double) * size_B;
    double* h_B;
    checkCudaErrors(hipHostMalloc(&h_B, mem_size_B));
    hipStream_t stream;

    ConstantInit(h_A, size_A);
    ConstantInit(h_B, size_B);

    // Allocate device memory
    double* d_A, * d_B, * d_C;

    // Allocate host matrix C
    dim3 dimsC(dimsB.x, 1, 1);
    unsigned int mem_size_C = dimsC.x * dimsC.y * sizeof(double);
    double* h_C;
    checkCudaErrors(hipHostMalloc(&h_C, mem_size_C));

    if (h_C == NULL) {
        fprintf(stderr, "Failed to allocate host matrix C!\n");
        exit(EXIT_FAILURE);
    }

    checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_A), mem_size_A));
    checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_B), mem_size_B));
    checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_C), mem_size_C));

    checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

    // copy host memory to device
    checkCudaErrors(hipMemcpyAsync(d_A, h_A, mem_size_A, hipMemcpyHostToDevice, stream));
    checkCudaErrors(hipMemcpyAsync(d_B, h_B, mem_size_B, hipMemcpyHostToDevice, stream));

    // Setup execution parameters
    dim3 threads(1024);
    dim3 grid(1);

    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    checkCudaErrors(hipEventRecord(start, stream));

    MatrixMulCUDA <<< grid, threads, 0, stream >>> (d_C, d_A, d_B);

    checkCudaErrors(hipEventRecord(stop, stream));

    // Wait for the stop event to complete
    checkCudaErrors(hipEventSynchronize(stop));

    float msecTotal = 0.0f;
    checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

    printf("Total time in msec: %f\n", msecTotal);

    // Copy result from device to host
    checkCudaErrors(hipMemcpyAsync(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost, stream));
    checkCudaErrors(hipStreamSynchronize(stream));

    // Clean up memory
    checkCudaErrors(hipHostFree(h_A));
    checkCudaErrors(hipHostFree(h_B));
    checkCudaErrors(hipHostFree(h_C));
    checkCudaErrors(hipFree(d_A));
    checkCudaErrors(hipFree(d_B));
    checkCudaErrors(hipFree(d_C));
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));
    return 0;
}


int main(int argc, char** argv) {

    // This will pick the best possible CUDA capable device, otherwise
    // override the device ID based on input provided at the command line
    int dev = findCudaDevice(argc, (const char**)argv);

    int oneK = 1024;

    dim3 dimsA(oneK, oneK, 1);
    dim3 dimsB(oneK, 1, 1);

    int matrix_result = MatrixMultiply(dimsA, dimsB);

    exit(matrix_result);
}

